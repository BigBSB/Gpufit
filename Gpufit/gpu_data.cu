#include "hip/hip_runtime.h"
#include "gpu_data.cuh"
#include <hip/hip_runtime.h>
#include <>

GPUData::GPUData(Info const & info) :
    chunk_size_(0),
    info_(info),

    data_( info_.max_chunk_size_*info_.n_points_ ),
    weights_( info_.use_weights_ ? info_.n_points_ * info_.max_chunk_size_ : 0 ),
    parameters_( info_.max_chunk_size_*info_.n_parameters_ ),
    prev_parameters_( info_.max_chunk_size_*info_.n_parameters_ ),
    parameters_to_fit_indices_( info_.n_parameters_to_fit_ ),
    user_info_( info_.user_info_size_ ),

    chi_squares_( info_.max_chunk_size_ * info_.n_blocks_per_fit_),
    prev_chi_squares_( info_.max_chunk_size_ ),
    gradients_( info_.max_chunk_size_ * info_.n_parameters_to_fit_ * info_.n_blocks_per_fit_),
    hessians_( info_.max_chunk_size_ * info_.n_parameters_to_fit_ * info_.n_parameters_to_fit_ ),
    scaled_hessians_(info_.max_chunk_size_ * info_.n_parameters_to_fit_ * info_.n_parameters_to_fit_),
    deltas_( info_.max_chunk_size_ * info_.n_parameters_to_fit_ ),
    scaling_vectors_( info_.max_chunk_size_ * info_.n_parameters_to_fit_ ),

    values_( info_.max_chunk_size_ * info_.n_points_ ),
    derivatives_( info_.max_chunk_size_ * info_.n_points_ * info_.n_parameters_ ),
    temp_derivatives_( info_.max_chunk_size_ * info_.n_points_ * info_.n_parameters_ ),

    lambdas_( info_.max_chunk_size_ ),
    lambda_lower_bounds_( info_.max_chunk_size_ ),
    lambda_upper_bounds_( info_.max_chunk_size_ ),
    step_bounds_( info_.max_chunk_size_ ),
    actual_reductions_( info_.max_chunk_size_ ),
    predicted_reductions_( info_.max_chunk_size_ ),
    directive_derivatives_( info_.max_chunk_size_ ),
    approximation_ratios_( info_.max_chunk_size_ ),
    scaled_parameters_( info_.max_chunk_size_ * info_.n_parameters_ ),
    scaled_deltas_( info_.max_chunk_size_ * info_.n_parameters_ ),
    scaled_delta_norms_( info_.max_chunk_size_ ),
    phis_( info_.max_chunk_size_ ),
    phi_derivatives_( info_.max_chunk_size_ ),
    derivatives_delta_(info_.max_chunk_size_ * info_.n_points_),

    states_( info_.max_chunk_size_ ),
    finished_( info_.max_chunk_size_ ),
    iteration_failed_( info_.max_chunk_size_ ),
    lambda_accepted_( info_.max_chunk_size_ ),
    newton_step_accepted_( info_.max_chunk_size_ ),
    all_finished_( 1 ),
    all_lambdas_accepted_(1),
    n_iterations_( info_.max_chunk_size_ ),

    decomposed_hessians_( info_.max_chunk_size_ * info_.n_parameters_to_fit_ * info_.n_parameters_to_fit_ ),
    inverted_hessians_( info_.max_chunk_size_ * info_.n_parameters_to_fit_ * info_.n_parameters_to_fit_ ),
    pointer_decomposed_hessians_( info_.max_chunk_size_ ),
    pointer_inverted_hessians_( info_.max_chunk_size_ ),
    pointer_deltas_( info_.max_chunk_size_ ),
    pivot_vectors_( info_.max_chunk_size_ * info_.n_parameters_to_fit_ ),
    cublas_info_( info_.max_chunk_size_ )
{
    hipblasCreate(&cublas_handle_);
}

GPUData::~GPUData()
{
    hipblasDestroy(cublas_handle_);
}

void GPUData::init
(
    int const chunk_size,
    int const chunk_index,
    double const * const data,
    double const * const weights,
    double const * const initial_parameters,
    std::vector<int> const & parameters_to_fit_indices)
{
    chunk_size_ = chunk_size;
    chunk_index_ = chunk_index;

    set(prev_chi_squares_, 0., chunk_size_);
    set(states_, 0, chunk_size_);
    set(finished_, 0, chunk_size_);
    set(scaling_vectors_, 0., chunk_size_ * info_.n_parameters_to_fit_);

    write(
        data_,
        &data[chunk_index_*info_.max_chunk_size_*info_.n_points_],
        chunk_size_*info_.n_points_);
    
    if (info_.use_weights_)
        write(weights_, &weights[chunk_index_*info_.max_chunk_size_*info_.n_points_],
                chunk_size_*info_.n_points_);
    
    write(
        parameters_,
        &initial_parameters[chunk_index_*info_.max_chunk_size_*info_.n_parameters_],
        chunk_size_ * info_.n_parameters_);
    
    write(parameters_to_fit_indices_, parameters_to_fit_indices);

    set(lambdas_, 0., chunk_size_);
    set(lambda_accepted_, 0, chunk_size_);
    set(newton_step_accepted_, 1, chunk_size_);
}

void GPUData::init_user_info(char const * const user_info)
{
    if (info_.user_info_size_ > 0)
        write(user_info_, user_info, info_.user_info_size_);
}

void GPUData::read(bool * dst, int const * src)
{
    int int_dst = 0;
    CUDA_CHECK_STATUS(hipMemcpy(&int_dst, src, sizeof(int), hipMemcpyDeviceToHost));
    * dst = (int_dst == 1) ? true : false;
}

void GPUData::write(double* dst, double const * src, int const count)
{
    CUDA_CHECK_STATUS(hipMemcpy(dst, src, count * sizeof(double), hipMemcpyHostToDevice));
}

void GPUData::write(int* dst, std::vector<int> const & src)
{
    std::size_t const size = src.size() * sizeof(int);
    CUDA_CHECK_STATUS(hipMemcpy(dst, src.data(), size, hipMemcpyHostToDevice));
}

void GPUData::write(char* dst, char const * src, std::size_t const count)
{
    CUDA_CHECK_STATUS(hipMemcpy(dst, src, count * sizeof(char), hipMemcpyHostToDevice));
}

void GPUData::copy(double * dst, double const * src, std::size_t const count)
{
    CUDA_CHECK_STATUS(hipMemcpy(dst, src, count * sizeof(double), hipMemcpyDeviceToDevice));
}

__global__ void set_kernel(int* dst, int const value, int const count)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= count)
        return;

    dst[index] = value;
}

void GPUData::set(int* arr, int const value, int const count)
{
    int const tx = 256;
	int const bx = (count / tx) + 1;

    dim3  threads(tx, 1, 1);
    dim3  blocks(bx, 1, 1);

    set_kernel<<< blocks, threads >>>(arr, value, count);
    CUDA_CHECK_STATUS(hipGetLastError());
}

void GPUData::set(int* arr, int const value)
{
    int const tx = 1;
    int const bx = 1;

    dim3  threads(tx, 1, 1);
    dim3  blocks(bx, 1, 1);

    set_kernel<<< blocks, threads >>>(arr, value, 1);
    CUDA_CHECK_STATUS(hipGetLastError());
}

__global__ void set_kernel(double* dst, double const value, std::size_t const count)
{
	std::size_t const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= count)
        return;

    dst[index] = value;
}

void GPUData::set(double* arr, double const value, int const count)
{
    int const tx = 256;
	int const bx = (count / tx) + 1;

    dim3  threads(tx, 1, 1);
    dim3  blocks(bx, 1, 1);
    set_kernel<<< blocks, threads >>>(arr, value, count);
    CUDA_CHECK_STATUS(hipGetLastError());
}
